
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel for matrix multiplication
__global__ void MatrixMulKernel(float *A, float *B, float *C, int width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < width && Col < width) {
        float value = 0;
        for (int k = 0; k < width; ++k)
            value += A[Row * width + k] * B[k * width + Col];
        C[Row * width + Col] = value;
    }
}

int main() {
    int width = 512;  // matrix dimension (width x width)
    size_t size = width * width * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < width * width; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy matrices A and B to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int BLOCK_SIZE = 16;
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (width + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel
    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // (Optional) Verify result by printing a few values
    printf("C[0] = %f\n", h_C[0]);
    printf("C[last] = %f\n", h_C[width * width - 1]);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
