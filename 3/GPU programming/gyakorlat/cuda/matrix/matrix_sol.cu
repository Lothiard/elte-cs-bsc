
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_WIDTH 16  // Defines the size of each tile (submatrix)

// -----------------------------------------------------------------------------
// GPU Kernel: MatrixMulTiled
// Computes C = A � B using tiling and shared memory optimization.
// Each thread block computes one TILE_WIDTH � TILE_WIDTH submatrix of C.
// -----------------------------------------------------------------------------
__global__ void MatrixMulTiled(float *A, float *B, float *C, int width) {
    // Shared memory for storing tiles of A and B
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    // Compute global row and column index of the element to compute
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Accumulator for the final value of C[Row, Col]
    float value = 0.0f;

    // Loop over all tiles of A and B needed to compute one tile of C
    for (int t = 0; t < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {

        // ---------------------------------------------------------------------
        // Load one tile of A and one tile of B into shared memory.
        // Each thread loads one element from A and one from B.
        // Check for boundaries to avoid illegal memory access.
        // ---------------------------------------------------------------------
        if (Row < width && (t * TILE_WIDTH + threadIdx.x) < width)
            As[threadIdx.y][threadIdx.x] = A[Row * width + t * TILE_WIDTH + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;  // Pad with zero outside valid range

        if (Col < width && (t * TILE_WIDTH + threadIdx.y) < width)
            Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * width + Col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;  // Pad with zero outside valid range

        // Synchronize all threads in the block before using shared memory
        __syncthreads();

        // ---------------------------------------------------------------------
        // Perform partial computation for the tile of C.
        // Each thread computes one element of C's tile by multiplying
        // corresponding elements of the shared tiles As and Bs.
        // ---------------------------------------------------------------------
        for (int k = 0; k < TILE_WIDTH; ++k)
            value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

        // Synchronize again before loading next tile
        __syncthreads();
    }

    // -------------------------------------------------------------------------
    // Write the computed value to the output matrix (if within bounds)
    // -------------------------------------------------------------------------
    if (Row < width && Col < width)
        C[Row * width + Col] = value;
}

// -----------------------------------------------------------------------------
// Host Code: main()
// Allocates matrices, initializes them, runs the kernel, and verifies output.
// -----------------------------------------------------------------------------
int main() {
    int width = 512;                  // Matrix dimension (width x width)
    size_t size = width * width * sizeof(float);  // Memory size in bytes

    // -------------------------------------------------------------------------
    // Allocate memory on host (CPU)
    // -------------------------------------------------------------------------
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input matrices with random values
    for (int i = 0; i < width * width; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // -------------------------------------------------------------------------
    // Allocate memory on device (GPU)
    // -------------------------------------------------------------------------
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // -------------------------------------------------------------------------
    // Copy input matrices from host to device
    // -------------------------------------------------------------------------
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // -------------------------------------------------------------------------
    // Define CUDA grid and block dimensions
    // - Each block handles TILE_WIDTH � TILE_WIDTH elements of C
    // - The grid dimensions cover the entire output matrix
    // -------------------------------------------------------------------------
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((width + TILE_WIDTH - 1) / TILE_WIDTH,
                 (width + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    // -------------------------------------------------------------------------
    // Launch the kernel on the GPU
    // -------------------------------------------------------------------------
    MatrixMulTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    // Wait for GPU to finish before accessing results
    hipDeviceSynchronize();

    // -------------------------------------------------------------------------
    // Copy result matrix C back to host
    // -------------------------------------------------------------------------
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // -------------------------------------------------------------------------
    // Print sample elements from result for verification
    // -------------------------------------------------------------------------
    printf("Sample output:\n");
    printf("C[0] = %f\n", h_C[0]);
    printf("C[last] = %f\n", h_C[width * width - 1]);

    // -------------------------------------------------------------------------
    // Free device and host memory
    // -------------------------------------------------------------------------
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
